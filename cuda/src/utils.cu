#include "hip/hip_runtime.h"
#include "utils.cuh"

double get_sec()
{
    struct timeval time;
    gettimeofday(&time, NULL);
    return (1e6 * time.tv_sec + time.tv_usec);
}

double cpu_elapsed_time(double &beg, double &end) { return 1.0e-6 * (end - beg); }

void cudaCheck(hipError_t error, const char *file, int line)
{
    if (error != hipSuccess)
    {
        printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
               hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
};

void CudaDeviceInfo()
{
    int deviceId;

    hipGetDevice(&deviceId);

    hipDeviceProp_t props{};
    hipGetDeviceProperties(&props, deviceId);

    printf("Device ID: %d\n\
        Name: %s\n\
        Compute Capability: %d.%d\n\
        memoryBusWidth: %d\n\
        maxThreadsPerBlock: %d\n\
        maxThreadsPerMultiProcessor: %d\n\
        maxRegsPerBlock: %d\n\
        maxRegsPerMultiProcessor: %d\n\
        totalGlobalMem: %zuMB\n\
        sharedMemPerBlock: %zuKB\n\
        sharedMemPerMultiprocessor: %zuKB\n\
        totalConstMem: %zuKB\n\
        multiProcessorCount: %d\n\
        Warp Size: %d\n",
           deviceId, props.name, props.major, props.minor, props.memoryBusWidth,
           props.maxThreadsPerBlock, props.maxThreadsPerMultiProcessor,
           props.regsPerBlock, props.regsPerMultiprocessor,
           props.totalGlobalMem / 1024 / 1024, props.sharedMemPerBlock / 1024,
           props.sharedMemPerMultiprocessor / 1024, props.totalConstMem / 1024,
           props.multiProcessorCount, props.warpSize);
};

void randomize_matrix(double *mat, int N)
{
    // NOTICE: Use gettimeofday instead of srand((unsigned)time(NULL)); the time
    // precision is too low and the same random number is generated.
    struct timeval time
    {
    };
    gettimeofday(&time, nullptr);
    srand(time.tv_usec);
    for (int i = 0; i < N; i++)
    {
        double tmp = (double)(rand() % 5) + 0.01 * (rand() % 5);
        tmp = (rand() % 2 == 0) ? tmp : tmp * (-1.);
        mat[i] = tmp;
    }
}

void range_init_matrix(double *mat, int N)
{
    for (int i = 0; i < N; i++)
    {
        mat[i] = i;
    }
}

void zero_init_matrix(double *mat, int N)
{
    for (int i = 0; i < N; i++)
    {
        mat[i] = 0.0;
    }
}

void copy_matrix(const double *src, double *dest, int N)
{
    int i;
    for (i = 0; src + i && dest + i && i < N; i++)
        *(dest + i) = *(src + i);
    if (i != N)
        printf("copy failed at %d while there are %d elements in total.\n", i, N);
}

void print_matrix(const double *A, int M, int N, std::ofstream &fs)
{
    int i;
    fs << std::setprecision(2)
       << std::fixed; // Set doubleing-point precision and fixed notation
    fs << "[";
    for (i = 0; i < M * N; i++)
    {
        if ((i + 1) % N == 0)
            fs << std::setw(5) << A[i]; // Set field width and write the value
        else
            fs << std::setw(5) << A[i] << ", ";
        if ((i + 1) % N == 0)
        {
            if (i + 1 < M * N)
                fs << ";\n";
        }
    }
    fs << "]\n";
}

bool verify_matrix(double *matRef, double *matOut, int N)
{
    double diff = 0.0;
    int i;
    for (i = 0; i < N; i++)
    {
        diff = std::fabs(matRef[i] - matOut[i]);
        if (diff > 0.01)
        {
            printf("Divergence! Should %5.2f, Is %5.2f (Diff %5.2f) at %d\n",
                   matRef[i], matOut[i], diff, i);
            return false;
        }
    }
    return true;
}

int div_ceil(int numerator, int denominator)
{
    std::div_t res = std::div(numerator, denominator);
    return res.rem ? (res.quot + 1) : res.quot;
}

void run_kernel(int kernel_num, hipblasHandle_t handle, DGEMM_FUNC_SIGNITURE)
{
    switch (kernel_num)
    {
    case 0:
        dgemm_00_cublas(handle, DGEMM_FUNC_PARAM);
        break;
    case 1:
        dgemm_01_naive(DGEMM_FUNC_PARAM);
        break;
    case 2:
        dgemm_02_ideal(DGEMM_FUNC_PARAM);
        break;
    default:
        throw std::invalid_argument("Unknown kernel number");
    }
}