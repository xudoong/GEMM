#include "dgemm.cuh"


void dgemm_00_cublas(hipblasHandle_t handle, DGEMM_FUNC_SIGNITURE) {
    // cublas uses column-major order, while we use row-major order.
    // So we compute C^T=alpha * B^T * A^T + beta * C^T.
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, N,
        A, K, &beta, C, N);
}