#include "hip/hip_runtime.h"
#include "gemm.cuh"
#include <cuda/pipeline>
#include <hip/hip_cooperative_groups.h>

const int bM = 128;
const int bN = 128;
const int bK = 32;
const int wM = 64;
const int wN = 64;
const int tM = 16;
const int tN = 8;
const int tK = 16;

// number of pipeline stages
const int num_stages = 2;

const int num_wM = bM / wM;
const int num_wN = bN / wN;

const int num_tM = wM / tM;
const int num_tN = wN / tN;

const int nthreads = 32 * num_wM * num_wN;

// use for load global memory -> shared memory
const int ldWidth = 128 / 8 / sizeof(half);
const int bK2 = bK / ldWidth;
const int bN2 = bN / ldWidth;
const int stride_A_m = nthreads / bK2;
const int stride_B_k = nthreads / bN2;

#define FLOAT4(ptr) ((float4*)ptr)
#define CFLOAT4(ptr) ((const float4*)ptr)

// padding shared memory to avoid bank conflict
const int pad_half = 8;
const int pad_float4 = 1;

using pipeline_t = cuda::pipeline<cuda::thread_scope::thread_scope_block>;

__device__ static void load_shared_A(void *dst, const void *src, int K, pipeline_t &pipeline,
                                     int offset_A_m, int offset_A_k) {
    for (int i = 0; i < bM; i += stride_A_m) {
        int sAi = (i + offset_A_m) * (bK2 + pad_float4) + offset_A_k;
        int gAi = (i + offset_A_m) * (K / ldWidth) + offset_A_k;
        cuda::memcpy_async(&FLOAT4(dst)[sAi], &CFLOAT4(src)[gAi], sizeof(float4), pipeline);
    }
}

__device__ static void load_shared_B(void *dst, const void *src, int N, pipeline_t &pipeline,
                                     int offset_B_k, int offset_B_n) {
    for (int i = 0; i < bK; i += stride_B_k) {
        int sBi = (i + offset_B_k) * (bN2 + pad_float4) + offset_B_n;
        int gBi = (i + offset_B_k) * (N / ldWidth) + offset_B_n;
        cuda::memcpy_async(&FLOAT4(dst)[sBi], &CFLOAT4(src)[gBi], sizeof(float4), pipeline);
    }
}

template <typename T>
__device__ static void swap(T &a, T &b) {
    T tmp = a;
    a = b;
    b = tmp;
}

__device__ static void compute(const half *sA, const half *sB, uint32_t *rA, uint32_t *rB, float *rC) {
    const int lane = threadIdx.x;
    const int tAi = lane % 16;
    const int tAk = lane / 16;

// version: ldmatrix: TFLOPS 46.9%
    const int tBk = lane % 16;

// version: direct load: TFLOPS: 46.1%
//    const int tBk = lane / 4;
//    const int tBj = (lane % 4) * 2;

//  slow version: load B from shared memory -> register and then shuffle: TFLOPS < 20%
//  const int tBk = (lane % 4) * 2 + (lane / 4) % 2;
//  const int tBj = (lane / 8) * 2;
//  bool up = (lane / 4) % 2 == 0;

    const int wi = threadIdx.y;
    const int wj = threadIdx.z;

    const half *sAw = &sA[wi * wM * (bK + pad_half)];
    const half *sBw = &sB[wj * wN];

    uint32_t smem_ptr;

    for (int k = 0; k < bK; k += tK) {
        for (int j = 0; j < wN; j += tN) {
            // load B from shared memory -> register
            const half *sBt = &sBw[k * (bN + pad_half) + j];
            uint32_t *rBt = &rB[j / tN * 2];

// version: direct load
//            rBt[0] = *((uint32_t * )(&sBt[(tBk + 0) * (bN + pad_half) + tBj]));
//            rBt[1] = *((uint32_t * )(&sBt[(tBk + 8) * (bN + pad_half) + tBj]));
//            asm volatile ( "movmatrix.sync.aligned.m8n8.trans.b16 %0, %1;"
//                            : "=r"(rBt[0]) : "r"(rBt[0])
//                        );
//            asm volatile ( "movmatrix.sync.aligned.m8n8.trans.b16 %0, %1;"
//                    : "=r"(rBt[1]) : "r"(rBt[1])
//                    );

// version: ldmatrix
            smem_ptr = __cvta_generic_to_shared(&sBt[tBk * (bN + pad_half)]);
            asm volatile ( "ldmatrix.sync.aligned.x2.m8n8.shared.b16 "
                           "{%0, %1}, [%2]; "
                    : "=r"(rBt[0]), "=r"(rBt[1])
                    : "r"(smem_ptr)
                    );
            asm volatile ( "movmatrix.sync.aligned.m8n8.trans.b16 %0, %1;"
                            : "=r"(rBt[0]) : "r"(rBt[0])
                        );
            asm volatile ( "movmatrix.sync.aligned.m8n8.trans.b16 %0, %1;"
                    : "=r"(rBt[1]) : "r"(rBt[1])
                    );

//  slow version: load B from shared memory -> register and then shuffle
//            half *rBh = (half *)rBt;
//            swap(rBh[1], rBh[2]);
//            if (up) {
//                rBt[1] = __shfl_xor_sync(0xffffffff, rBt[1], 4);
//            } else {
//                rBt[0] = __shfl_xor_sync(0xffffffff, rBt[0], 4);
//            }
//            swap(rBh[1], rBh[2]);
        }

        for (int i = 0; i < wM; i += tM) {
            // load A from shared memory -> register
            smem_ptr = __cvta_generic_to_shared(&sAw[(i + tAi) * (bK + pad_half) + k + tAk * 8]);
            asm volatile ( "ldmatrix.sync.aligned.x4.m8n8.shared.b16 "
                           "{%0, %1, %2, %3}, [%4]; "
                           : "=r"(rA[0]), "=r"(rA[1]), "=r"(rA[2]), "=r"(rA[3])
                           : "r"(smem_ptr)
                           );
            for (int j = 0; j < wN; j += tN) {
                uint32_t *rBt = &rB[j / tN * 2];
                float *rCt = &rC[(i / tM * num_tN + j / tN) * 4];
                asm( "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 "
                     " { %0, %1, %2, %3 }, "
                     " { %4, %5, %6, %7 }, "
                     " { %8, %9 }, "
                     " { %10, %11, %12, %13 };"
                     : "=f"(rCt[0]), "=f"(rCt[1]), "=f"(rCt[2]), "=f"(rCt[3])
                     : "r"(rA[0]), "r"(rA[1]), "r"(rA[2]), "r"(rA[3]),
                       "r"(rBt[0]), "r"(rBt[1]),
                       "f"(rCt[0]), "f"(rCt[1]), "f"(rCt[2]), "f"(rCt[3])
               );
            }
        }
    }
}

__global__ static void kernel(GEMM_FUNC_SIGNITURE) {
    __shared__ half sA[num_stages][bM * (bK + pad_half)];
    __shared__ half sB[num_stages][bK * (bN + pad_half)];

    // m16n8k16 mma
    uint32_t rA[4];
    uint32_t rB[num_tN * 2];
    float rC[num_tM * num_tN * 4] = {0};

    int bi = blockIdx.x;
    int bj = blockIdx.y;

    A += bi * bM * K;
    B += bj * bN;
    C += bi * bM * N + bj * bN;

    const int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;

    const int wi = threadIdx.y;
    const int wj = threadIdx.z;

    C += wi * wM * N + wj * wN;

    const int offset_A_m = tid / bK2;
    const int offset_A_k = tid % bK2;

    const int offset_B_k = tid / bN2;
    const int offset_B_n = tid % bN2;

    __shared__ cuda::pipeline_shared_state<cuda::thread_scope::thread_scope_block, num_stages> shared_state;
    auto pipeline = cuda::make_pipeline(cooperative_groups::this_thread_block(), &shared_state);

    for (int stage = 0; stage < num_stages - 1; stage++) {
        pipeline.producer_acquire();
        load_shared_A(sA[stage], A, K, pipeline, offset_A_m, offset_A_k);
        load_shared_B(sB[stage], B, N, pipeline, offset_B_k, offset_B_n);
        pipeline.producer_commit();
        A += bK;
        B += bK * N;
    }

    for (int s = num_stages - 1; s < K / bK; s++) {
        int comp_stage = (s - num_stages + 1) % num_stages;
        int copy_stage = s % num_stages;

        pipeline.producer_acquire();
        load_shared_A(sA[copy_stage], A, K, pipeline, offset_A_m, offset_A_k);
        load_shared_B(sB[copy_stage], B, N, pipeline, offset_B_k, offset_B_n);
        pipeline.producer_commit();

        pipeline.consumer_wait();
        compute(sA[comp_stage], sB[comp_stage], rA, rB, rC);
        pipeline.consumer_release();

        // advance to next tile
        A += bK;
        B += bK * N;
    }

    for (int s = 0; s < num_stages - 1; s++) {
        int stage = (K / bK + 1 + s) % num_stages;
        pipeline.consumer_wait();
        compute(sA[stage], sB[stage], rA, rB, rC);
        pipeline.consumer_release();
    }

    // write to C
    const int tCi = threadIdx.x / 4;
    const int tCj = threadIdx.x % 4;
    for (int i = 0; i < wM; i += tM) {
        for (int j = 0; j < wN; j += tN) {
            float *rCt = &rC[(i / tM * num_tN + j / tN) * 4];
            C[(i + tCi + 0) * N + j + tCj * 2 + 0] = rCt[0];
            C[(i + tCi + 0) * N + j + tCj * 2 + 1] = rCt[1];
            C[(i + tCi + 8) * N + j + tCj * 2 + 0] = rCt[2];
            C[(i + tCi + 8) * N + j + tCj * 2 + 1] = rCt[3];
        }
    }
}

void gemm_06_mma(GEMM_FUNC_SIGNITURE) {
    assert(M % bM == 0);
    assert(N % bN == 0);
    assert(K % bK == 0);
    if (alpha != 1 || beta != 0) {
        std::cout << "gemm_06_mma kernel only supports computing C=A*B (alpha=1, beta=0)\n";
        exit(-1);
    }

    dim3 gridDim(M / bM, N / bN);
    dim3 blockDim(32, num_wM, num_wN);
    kernel<<<gridDim, blockDim>>>(GEMM_FUNC_PARAM);
}