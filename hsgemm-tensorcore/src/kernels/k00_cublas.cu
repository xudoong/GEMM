#include "hip/hip_runtime.h"
#include "gemm.cuh"


void gemm_00_cublas(hipblasHandle_t handle, GEMM_FUNC_SIGNITURE) {
    // cublas uses column-major order, while we use row-major order.
    // So we compute C^T=alpha * B^T * A^T + beta * C^T.
    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                N, M, K,
                &alpha,
                reinterpret_cast<const __half*>(B), HIP_R_16F, N,
                reinterpret_cast<const __half*>(A), HIP_R_16F, K,
                &beta,
                reinterpret_cast<       float*>(C), HIP_R_32F, N,
                HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT);
}