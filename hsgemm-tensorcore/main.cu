#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <fstream>
#include <iostream>
#include <vector>

#include "utils.cuh"

#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

const std::string errLogFile = "matrixValidationFailure.txt";

const int theoretical_max_tflops = 312;
int default_size = 4096;
int repeat_times = 50;
int n_warmup = 5;

int main(int argc, char **argv) {
    if (argc < 2) {
        std::cerr << "Usage: <this-exe> <kernel-number> [[m <m_size>] [n <n_size>] [k <k_size>] [r <repeat_times>]" << std::endl;
        exit(EXIT_FAILURE);
    }

    int m, n, k;
    m = n = k = default_size;
    if (argc > 2) {
        for (int i = 2; i < argc; i++) {
            if (argv[i][0] == 'm' && i < argc - 1) {
                m = std::stoi(argv[i + 1]);
            }
            else if (argv[i][0] == 'n' && i < argc - 1) {
                n = std::stoi(argv[i + 1]);
            }
            else if (argv[i][0] == 'k' && i < argc - 1) {
                k = std::stoi(argv[i + 1]);
            }
            else if (argv[i][0] == 'r' && i < argc - 1) {
                repeat_times = std::stoi(argv[i + 1]);
            }
        }
    }

    // get kernel number
    int kernel_num = std::stoi(argv[1]);

    // get environment variable for device
    int deviceIdx = 0;
    if (getenv("DEVICE") != NULL) {
        deviceIdx = atoi(getenv("DEVICE"));
    }
    cudaCheck(hipSetDevice(deviceIdx));

    printf("Running kernel %d on device %d.\n", kernel_num, deviceIdx);

    // print some device info
    // CudaDeviceInfo();

    // Declare the handle, create the handle, hipblasCreate will return a value of
    // type hipblasStatus_t to determine whether the handle was created
    // successfully (the value is 0)
    hipblasHandle_t handle;
    if (hipblasCreate(&handle)) {
        std::cerr << "Create cublas handle error." << std::endl;
        exit(EXIT_FAILURE);
    };

    // Using cudaEvent for gpu stream timing, cudaEvent is equivalent to
    // publishing event tasks in the target stream
    hipEvent_t beg, end;
    hipEventCreate(&beg);
    hipEventCreate(&end);


//    float alpha = 0.5, beta = 3.0; // GEMM input parameters, C=α*AB+β*C
    float alpha = 1, beta = 0; // GEMM input parameters, C=α*AB+β*C

    half *A = nullptr;
    half *B = nullptr;
    float *C = nullptr;
    float *C_ref = nullptr;
    half *dA = nullptr;
    half *dB = nullptr;
    float *dC = nullptr;
    float *dC_ref = nullptr;

    int max_size = std::max(m, n);
    max_size = std::max(max_size, k);
    A = (half *) malloc(sizeof(half) * max_size * max_size);
    B = (half *) malloc(sizeof(half) * max_size * max_size);
    C = (float *) malloc(sizeof(float) * max_size * max_size);
    C_ref = (float *) malloc(sizeof(float) * max_size * max_size);

    randomize_matrix(A, max_size * max_size);
    randomize_matrix(B, max_size * max_size);
    randomize_matrix(C, max_size * max_size);

    cudaCheck(hipMalloc((void **) &dA, sizeof(half) * max_size * max_size));
    cudaCheck(hipMalloc((void **) &dB, sizeof(half) * max_size * max_size));
    cudaCheck(hipMalloc((void **) &dC, sizeof(float) * max_size * max_size));
    cudaCheck(hipMalloc((void **) &dC_ref, sizeof(float) * max_size * max_size));

    cudaCheck(hipMemcpy(dA, A, sizeof(half) * max_size * max_size,
                         hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dB, B, sizeof(half) * max_size * max_size,
                         hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dC, C, sizeof(float) * max_size * max_size,
                         hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dC_ref, C, sizeof(float) * max_size * max_size,
                         hipMemcpyHostToDevice));

    printf("Size: (m, n, k) = (%d, %d, %d).\n", m, n, k);
    printf("Repeat %d times.\n", repeat_times);
    // Verify the correctness of the calculation, and execute it once before the
    // kernel function timing to avoid cold start errors
    if (kernel_num != 0 && kernel_num != FAKE_KERNEL_NUMBER) {
        run_kernel(0, handle, m, n, k, alpha, dA, dB, beta, dC_ref);      // cuBLAS
        run_kernel(kernel_num, handle, m, n, k, alpha, dA, dB, beta,
                   dC); // Executes the kernel, modifies the result matrix
        cudaCheck(hipDeviceSynchronize());
        cudaCheck(hipGetLastError()); // Check for async errors during kernel run
        hipMemcpy(C, dC, sizeof(float) * m * n, hipMemcpyDeviceToHost);
        hipMemcpy(C_ref, dC_ref, sizeof(float) * m * n, hipMemcpyDeviceToHost);

        if (!verify_matrix(C_ref, C, m * n)) {
            std::cout
                    << "Failed to pass the correctness verification against NVIDIA "
                       "cuBLAS."
                    << std::endl;
            if (m <= 128) {
                std::cout << " Logging faulty output into " << errLogFile << "\n";
                std::ofstream fs;
                fs.open(errLogFile);
                fs << "A:\n";
                print_matrix(A, m, n, fs);
                fs << "B:\n";
                print_matrix(B, m, n, fs);
                fs << "C:\n";
                print_matrix(C, m, n, fs);
                fs << "Should:\n";
                print_matrix(C_ref, m, n, fs);
            }
            exit(EXIT_FAILURE);
        }
    }

    // warmup
    for (int j = 0; j < n_warmup; j++) {
        run_kernel(kernel_num, handle, m, n, k, alpha, dA, dB, beta, dC);
    }

    // benchmark region
    std::vector<float> elapsed_times;
    for (int j = 0; j < repeat_times; j++) {
        float elapsed_time;
        hipEventRecord(beg);
        // We don't reset dC between runs to save time
        run_kernel(kernel_num, handle, m, n, k, alpha, dA, dB, beta, dC);
        hipEventRecord(end);
        hipEventSynchronize(beg);
        hipEventSynchronize(end);
        hipEventElapsedTime(&elapsed_time, beg, end);
        elapsed_times.push_back(elapsed_time / 1000);  // Convert to seconds
    }
    // report performance

    std::sort(elapsed_times.begin(), elapsed_times.end());
    std::reverse(elapsed_times.begin(), elapsed_times.end());
    float tile05_time = elapsed_times[int(repeat_times * 0.05)];
    float tile50_time = elapsed_times[int(repeat_times * 0.50)];
    float tile95_time = elapsed_times[int(repeat_times * 0.95)];

    float flops = 2.0 * float(m) * float(n) * k;

    float tflops05 = flops / tile05_time / 1e12;
    float tflops50 = flops / tile50_time / 1e12;
    float tflops95 = flops / tile95_time / 1e12;

    printf("TFLOPS: %5.1f (5%%)  %5.1f (50%%)  %5.1f (95%%). %.1f%% of theoretical.\n",
           tflops05, tflops50, tflops95, tflops95 / theoretical_max_tflops * 100);
    // make dC and dC_ref equal again (we modified dC while calling our kernel
    // for benchmarking)
    cudaCheck(hipMemcpy(dC, dC_ref, sizeof(float) * m * n,
                         hipMemcpyDeviceToDevice));

    // Free up CPU and GPU space
    free(A);
    free(B);
    free(C);
    free(C_ref);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipFree(dC_ref);
    hipblasDestroy(handle);

    return 0;
};