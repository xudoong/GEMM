#include "gemm.cuh"


void gemm_00_cublas(hipblasHandle_t handle, GEMM_FUNC_SIGNITURE) {
    // cublas uses column-major order, while we use row-major order.
    // So we compute C^T=alpha * B^T * A^T + beta * C^T.
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, N,
        A, K, &beta, C, N);
}