#include "gemm.cuh"


void gemm_00_cublas(hipblasHandle_t handle, GEMM_FUNC_SIGNITURE) {
    // cublas uses column-major order, while we use row-major order.
    // So we compute C^T=alpha * B * A^T + beta * C^T.
    hipblasHgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                N, M, K,
                &alpha,
                B, K,
                A, K,
                &beta,
                C, N);
}