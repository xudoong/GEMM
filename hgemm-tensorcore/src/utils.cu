#include "hip/hip_runtime.h"
#include "utils.cuh"

double get_sec()
{
    struct timeval time;
    gettimeofday(&time, NULL);
    return (1e6 * time.tv_sec + time.tv_usec);
}

double cpu_elapsed_time(double &beg, double &end) { return 1.0e-6 * (end - beg); }

void cudaCheck(hipError_t error, const char *file, int line)
{
    if (error != hipSuccess)
    {
        printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
               hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
};

void CudaDeviceInfo()
{
    int deviceId;

    hipGetDevice(&deviceId);

    hipDeviceProp_t props{};
    hipGetDeviceProperties(&props, deviceId);

    printf("Device ID: %d\n\
        Name: %s\n\
        Compute Capability: %d.%d\n\
        memoryBusWidth: %d\n\
        maxThreadsPerBlock: %d\n\
        maxThreadsPerMultiProcessor: %d\n\
        maxRegsPerBlock: %d\n\
        maxRegsPerMultiProcessor: %d\n\
        totalGlobalMem: %zuMB\n\
        sharedMemPerBlock: %zuKB\n\
        sharedMemPerMultiprocessor: %zuKB\n\
        totalConstMem: %zuKB\n\
        multiProcessorCount: %d\n\
        Warp Size: %d\n",
           deviceId, props.name, props.major, props.minor, props.memoryBusWidth,
           props.maxThreadsPerBlock, props.maxThreadsPerMultiProcessor,
           props.regsPerBlock, props.regsPerMultiprocessor,
           props.totalGlobalMem / 1024 / 1024, props.sharedMemPerBlock / 1024,
           props.sharedMemPerMultiprocessor / 1024, props.totalConstMem / 1024,
           props.multiProcessorCount, props.warpSize);
};

bool verify_matrix(half *matRef, half *matOut, int N)
{
    double diff = 0.0;
    int i;
    for (i = 0; i < N; i++)
    {
        diff = fabs(matRef[i] - matOut[i]);
        float max_abs = std::max(fabs(matRef[i]), fabs(matOut[i]));
        auto rerr = diff / (max_abs + 1e-8);
        if (diff > 0.1 || rerr > 0.1 || std::isnan(float(matOut[i])))
        {
            printf("Divergence! Should %5.2f, Is %5.2f (diff %5.2f, rerr %5.2f) at %d\n",
                   matRef[i], matOut[i], diff, rerr, i);
            return false;
        }
    }
    return true;
}

int div_ceil(int numerator, int denominator)
{
    std::div_t res = std::div(numerator, denominator);
    return res.rem ? (res.quot + 1) : res.quot;
}

void run_kernel(int kernel_num, hipblasHandle_t handle, GEMM_FUNC_SIGNITURE)
{
    switch (kernel_num)
    {
        case 0:
            gemm_00_cublas(handle, GEMM_FUNC_PARAM);
            break;
        case 1:
            gemm_01_fake(GEMM_FUNC_PARAM);
            break;
        case 2:
            gemm_02_naive(GEMM_FUNC_PARAM);
            break;
        case 3:
            gemm_03_wmma_shmem(GEMM_FUNC_PARAM);
            break;
        case 4:
            gemm_04_wmma_shmem_opt(GEMM_FUNC_PARAM);
            break;
       case 5:
            gemm_05_wmma_stage(GEMM_FUNC_PARAM);
            break;
       case 6:
            gemm_06_wmma_stage_dbreg(GEMM_FUNC_PARAM);
            break;
        default:
            std::cout << "Error: invalid kernel number " << kernel_num << std::endl;
            throw std::invalid_argument("Unknown kernel number");
    }
}